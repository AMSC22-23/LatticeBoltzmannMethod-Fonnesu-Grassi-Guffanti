#include "hip/hip_runtime.h"
// =========== STL INCLUDES ===========
#include <vector>
#include <array>
#include <cassert>
#include <functional>
// ======================================

// =========== EIGEN INCLUDES ===========
#include <Eigen/Core>
#include <Eigen/Dense>
#include <unsupported/Eigen/CXX11/Tensor>
// ======================================

// =========== CUDA INCLUDES ===========
#include <hip/hip_runtime.h> 
#include <>
// ======================================

__global__ void update_macro_kernel(const double* populations, const Point<2>* fluid_nodes, 
                                    double* global_rho, double* global_u, int num_fluid_nodes, int width, int height) {
    int fnode = blockIdx.x * blockDim.x + threadIdx.x;

    if (fnode < num_fluid_nodes) {

        int i = fluid_nodes[fnode].coords[0];
        int j = fluid_nodes[fnode].coords[1];

        int idx = i * height + j;

        double p0 = populations[idx * 9 + 0];
        double p1 = populations[idx * 9 + 1];
        double p2 = populations[idx * 9 + 2];
        double p3 = populations[idx * 9 + 3];
        double p4 = populations[idx * 9 + 4];
        double p5 = populations[idx * 9 + 5];
        double p6 = populations[idx * 9 + 6];
        double p7 = populations[idx * 9 + 7];
        double p8 = populations[idx * 9 + 8];

        double rho = p0 + p1 + p2 + p3 + p4 + p5 + p6 + p7 + p8;
        double rhoinv = 1.0 / rho;

        double ux = rhoinv * (p1 + p5 + p8 - (p3 + p6 + p7));
        double uy = rhoinv * (p2 + p5 + p6 - (p4 + p7 + p8));

        global_rho[idx] = rho;  
        global_u[idx * 2 + 0] = ux;
        global_u[idx * 2 + 1] = uy;
    }
}

void update_macro(const Tensor<double, 3> &populations, const std::vector<Point<2>> &fluid_nodes, Tensor<double, 2> &global_rho, Tensor<double, 3> &global_u)
{    

    size_t num_fluid_nodes = fluid_nodes.size();
    size_t width = populations.dimension(0);
    size_t height = populations.dimension(1);
    
    // Aloocation of the memory on the GPU
    double *d_populations, *d_global_rho, *d_global_u;
    Point<2>* d_fluid_nodes;

    hipMalloc(&d_populations, width * height * 9 * sizeof(double));
    hipMalloc(&d_global_rho, width * height * sizeof(double));
    hipMalloc(&d_global_u, width * height * 2 * sizeof(double));
    hipMalloc(&d_fluid_nodes, num_fluid_nodes * sizeof(Point<2>));

    // Copy data from CPU to GPU
    hipMemcpy(d_populations, populations.data(), width * height * 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_fluid_nodes, fluid_nodes.data(), num_fluid_nodes * sizeof(Point<2>), hipMemcpyHostToDevice);

    // Kernel size
    int blockSize = 256;
    int numBlocks = (num_fluid_nodes + blockSize - 1) / blockSize;

    // Kernel call
    update_macro_kernel<<<numBlocks, blockSize>>>(d_populations, d_fluid_nodes, d_global_rho, d_global_u, num_fluid_nodes, width, height);

    // Check for errors in the kernel
    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        std::cerr << "Errore nel kernel: " << hipGetErrorString(kernelErr) << std::endl;
        hipFree(d_populations);
        hipFree(d_fluid_nodes);
        hipFree(d_global_rho);
        hipFree(d_global_u);
        return;
    }

    // Copy of results from GPU to CPU
    hipMemcpy(global_rho.data(), d_global_rho, width * height * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(global_u.data(), d_global_u, width * height * 2 * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_populations);
    hipFree(d_global_rho);
    hipFree(d_global_u);
    hipFree(d_fluid_nodes);
}